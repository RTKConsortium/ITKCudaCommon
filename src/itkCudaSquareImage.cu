#include "hip/hip_runtime.h"
/*=========================================================================
 *
 *  Copyright NumFOCUS
 *
 *  Licensed under the Apache License, Version 2.0 (the "License");
 *  you may not use this file except in compliance with the License.
 *  You may obtain a copy of the License at
 *
 *         https://www.apache.org/licenses/LICENSE-2.0.txt
 *
 *  Unless required by applicable law or agreed to in writing, software
 *  distributed under the License is distributed on an "AS IS" BASIS,
 *  WITHOUT WARRANTIES OR CONDITIONS OF ANY KIND, either express or implied.
 *  See the License for the specific language governing permissions and
 *  limitations under the License.
 *
 *=========================================================================*/
#include "itkCudaSquareImage.hcu"
#include "CudaCommonExport.h"

namespace itk
{

template <class PixelType>
__global__ void
CudaSquareImage3D_kernel(int3 imSize, PixelType * in, PixelType * out)
{
  unsigned int i = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int j = blockIdx.y * blockDim.y + threadIdx.y;
  unsigned int k = blockIdx.z * blockDim.z + threadIdx.z;

  if (i >= imSize.x || j >= imSize.y || k >= imSize.z)
  {
    return;
  }
  unsigned int gidx = i + j * imSize.x + k * imSize.x * imSize.y;

  out[gidx] = in[gidx] * in[gidx];
}

template <class PixelType>
void
CudaSquareImage3D(int imSize[3], PixelType * in, PixelType * out)
{
  // Thread Block Dimensions
  constexpr int tBlock_x = 16;
  constexpr int tBlock_y = 4;
  constexpr int tBlock_z = 4;

  unsigned int blocksInX = (imSize[0] - 1) / tBlock_x + 1;
  unsigned int blocksInY = (imSize[1] - 1) / tBlock_y + 1;
  unsigned int blocksInZ = (imSize[2] - 1) / tBlock_z + 1;

  // Compute block and grid sizes
  dim3 dimGrid = dim3(blocksInX, blocksInY, blocksInZ);
  dim3 dimBlock = dim3(tBlock_x, tBlock_y, tBlock_z);

  int3 imageSize = make_int3(imSize[0], imSize[1], imSize[2]);

  CudaSquareImage3D_kernel<PixelType><<<dimGrid, dimBlock>>>(imageSize, in, out);
}

template void CudaCommon_EXPORT
CudaSquareImage3D<float>(int imSize[3], float * in, float * out);
template void CudaCommon_EXPORT
CudaSquareImage3D<double>(int imSize[3], double * in, double * out);

} // end namespace itk
